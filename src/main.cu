// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2015/2016, March 15 - April 15
// ###
// ###

#include "testingGPU.h"
#include <iostream>
#include <stdio.h>
#include <cstring>

#define DIM_C_REF 5

using namespace std;

int main(int argc, char **argv) {
  // Before the GPU can process your kernels, a so called "CUDA context" must be
  // initialized
  // This happens on the very first call to a CUDA function, and takes some time
  // (around half a second)
  // We will do it right here, so that the run time measurements are accurate
  hipDeviceSynchronize();
  CUDA_CHECK;

  // Reading command line parameters:
  // getParam("param", var, argc, argv) looks whether "-param xyz" is specified,
  // and if so stores the value "xyz" in "var"
  // If "-param" is not specified, the value of "var" remains unchanged
  //
  // return value: getParam("param", ...) returns true if "-param" is specified,
  // and false otherwise

  // input template image
  bool ret;
  string templateStr = "";
  ret = getParam("t", templateStr, argc, argv);
  if (!ret) cerr << "ERROR: no template image specified" << endl;

  // input observation image
  string observationStr = "";
  ret = getParam("o", observationStr, argc, argv);
  if (!ret) cerr << "ERROR: no observation image specified" << endl;

  // show the usage instructions
  if (argc <= 3) {
    cout << "Usage: " << argv[0] << " -t <template> -o <observation>" << endl;
    return 1;
  }

  // Load the input image using opencv (load as "grayscale", since we are
  // working only with binary shapes of single channel)
  cv::Mat observationIn =
      cv::imread(observationStr.c_str(), CV_LOAD_IMAGE_GRAYSCALE);
  // check
  if (observationIn.data == NULL) {
    cerr << "ERROR: Could not load observation image " << observationStr
         << endl;
    return 1;
  }

  cv::Mat templateIn = cv::imread(templateStr.c_str(), CV_LOAD_IMAGE_GRAYSCALE);
  // check
  if (templateIn.data == NULL) {
    cerr << "ERROR: Could not load template image " << templateStr << endl;
    return 1;
  }

  // convert to float representation (opencv loads image values as single bytes
  // by default)
  templateIn.convertTo(templateIn, CV_32F);
  // convert range of each channel to [0,1] (opencv default is [0,255])
  templateIn /= 255.f;
  // get image dimensions
  int t_w = templateIn.cols;  // width
  int t_h = templateIn.rows;  // height
  cout << "template image: " << t_w << " x " << t_h << endl;

  observationIn.convertTo(observationIn, CV_32F);
  // convert range of each channel to [0,1] (opencv default is [0,255])
  observationIn /= 255.f;
  // get image dimensions
  int o_w = observationIn.cols;  // width
  int o_h = observationIn.rows;  // height

  cout << "observation image: " << o_w << " x " << o_h << endl;

  // Set the output image format
  cv::Mat mOut(o_h, o_w, CV_32FC1);  // mOut will be a grayscale image, 1 layer
  // ### Define your own output images here as needed

  // Allocate arrays
  // input/output image width: w
  // input/output image height: h

  // allocate raw input image array
  float *observationImg = new float[(size_t)o_w * o_h];
  float *templateImg = new float[(size_t)t_w * t_h];

  // allocate raw output array (the computation result will be stored in this
  // array, then later converted to mOut for displaying)
  float *imgOut = new float[(size_t)o_w * o_h];

  // Init raw input image array
  // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
  // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
  // So we will convert as necessary, using interleaved "cv::Mat" for
  // loading/saving/displaying, and layered "float*" for CUDA computations

  convert_mat_to_layered(templateImg, templateIn);
  convert_mat_to_layered(observationImg, observationIn);

  Timer timer;
  timer.start();
  testALLGPU(templateImg, templateIn, observationImg, observationIn, t_w, t_h, o_w, o_h, imgOut) ;

  //stop timer here
  timer.end();
  float t = timer.get();  // elapsed time in seconds
  cout << "time: " << t * 1000 << " ms" << endl;

  // wait for key inputs
  cv::waitKey(0);
  // save input and result
  /*cv::imwrite("image_input.png",*/
  /*mIn * 255.f);  // "imwrite" assumes channel range [0,255]*/
  /*cv::imwrite("image_result.png", mOut * 255.f);*/

  // free allocated arrays
/*  delete[] observationImg;
  delete[] templateImg;

  delete[] imgOut;
  delete[] resizedImOut;*/

  // close all opencv windows
  cvDestroyAllWindows();
  return 0;
}
