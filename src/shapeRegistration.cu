#include "hip/hip_runtime.h"
/**
 *  \brief     Shape Registration code
 *  \details   Functions for Nonlinear Shape Registration without
 * Correspondences
 *  \author    Gerasimos Chourdakis
 *  \author    Sungjae Jung
 *  \author    Walter Simson
 *  \version   1.0cse
 *  \date      Mar. 2016
 *  \pre       to be added
 *  \bug       to be added
 *  \warning   to be added
 *  \copyright to be added
 */

#include "shapeRegistration.h"
#include <stdio.h>

void setQuadCoords(QuadCoords *qCoords, size_t w, size_t h) {
  for (size_t y = 0; y < h; y++) {
    for (size_t x = 0; x < w; x++) {
      qCoords->x[0] = (float)x - 0.5;
      qCoords->y[0] = (float)y - 0.5;
      qCoords->x[1] = (float)x + 0.5;
      qCoords->y[1] = (float)y - 0.5;
      qCoords->x[2] = (float)x + 0.5;
      qCoords->y[2] = (float)y + 0.5;
      qCoords->x[3] = (float)x - 0.5;
      qCoords->y[3] = (float)y + 0.5;
    }
  }
}

void cutMargins(float *imgIn, size_t w, size_t h, float *&resizedImg,
                int &resizedW, int &resizedH) {
  int top = -1;
  int bottom = -1;
  int left = -1;
  int right = -1;

  /** set the y-coordinate on the top of the image */
  for (size_t y = 0; y < h; y++) {
    for (size_t x = 0; x < w; x++) {
      if (imgIn[x + (w * y)] == FOREGROUND) {
        top = y;
        break;
      }
    }
    if (top != -1) {
      break;
    }
  }

  /** set the y-coordinate on the bottom of the image */
  for (size_t y = h; y > 0; y--) {
    for (size_t x = 0; x < w; x++) {
      if (imgIn[x + (w * y)] == FOREGROUND) {
        bottom = y;
        break;
      }
    }
    if (bottom != -1) {
      break;
    }
  }

  /** set the x-coordinate on the left of the image */
  for (size_t x = 0; x < w; x++) {
    for (size_t y = 0; y < h; y++) {
      if (imgIn[x + (w * y)] == FOREGROUND) {
        left = x;
        break;
      }
    }
    if (left != -1) {
      break;
    }
  }

  /** set the x-coordinate on the right of the image */
  for (size_t x = w; x > 0; x--) {
    for (size_t y = 0; y < h; y++) {
      if (imgIn[x + (w * y)] == FOREGROUND) {
        right = x;
        break;
      }
    }
    if (right != -1) {
      break;
    }
  }

  resizedH = bottom - top + 1;
  resizedW = right - left + 1;

  /** allocate raw input image array */
  resizedImg = new float[resizedW * resizedH];

  for (int y = 0; y < resizedH; y++) {
    for (int x = 0; x < resizedW; x++) {
      resizedImg[x + (size_t)(resizedW * y)] =
          imgIn[(x + left) + (w * (y + top))];
    }
  }
}

void centerOfMass(float *imgIn, size_t w, size_t h, float &xCentCoord,
                  float &yCentCoord) {
  int numOfForegroundPixel;

  xCentCoord = 0;
  yCentCoord = 0;
  numOfForegroundPixel = 0;

  for (size_t y = 0; y < h; y++) {
    for (size_t x = 0; x < w; x++) {
      if (imgIn[x + (w * y)] == FOREGROUND) {
        xCentCoord = xCentCoord + x;
        yCentCoord = yCentCoord + y;
        numOfForegroundPixel++;
      }
    }
  }

  xCentCoord /= numOfForegroundPixel;
  yCentCoord /= numOfForegroundPixel;
}

void imgNormalization(float *imgIn, size_t w, size_t h, QuadCoords *qCoords,
                      float xCentCoord, float yCentCoord) {
  /** NOTE: check max(xCentCoord, w - xCentCoord) again */
  float normXFactor = 0.5 / max(xCentCoord, w - xCentCoord);
  float normYFactor = 0.5 / max(yCentCoord, h - yCentCoord);

  size_t index;

  for (size_t y = 0; y < h; y++) {
    for (size_t x = 0; x < w; x++) {
      index = x + (w * y);
      qCoords[index].x[0] = (qCoords[index].x[0] - xCentCoord) * normXFactor;
      qCoords[index].x[1] = (qCoords[index].x[1] - xCentCoord) * normXFactor;
      qCoords[index].x[2] = (qCoords[index].x[2] - xCentCoord) * normXFactor;
      qCoords[index].x[3] = (qCoords[index].x[3] - xCentCoord) * normXFactor;

      qCoords[index].y[0] = (qCoords[index].y[0] - yCentCoord) * normYFactor;
      qCoords[index].y[1] = (qCoords[index].y[1] - yCentCoord) * normYFactor;
      qCoords[index].y[2] = (qCoords[index].y[2] - yCentCoord) * normYFactor;
      qCoords[index].y[3] = (qCoords[index].y[3] - yCentCoord) * normYFactor;
    }
  }
}

void imageMoment(float *imgIn, size_t w, size_t h, float *mmt,
                 size_t mmtDegree) {
  for (size_t p = 0; p < mmtDegree; p++) {
    for (size_t q = 0; q < mmtDegree; q++) {
      mmt[p + p * q] = 0;

      for (size_t y = 0; y < h; y++) {
        for (size_t x = 0; x < w; x++) {
          /** note: (q+p)th order in the dissertation but not here,
           *  need to check later
           */
          mmt[p + (p * q)] +=
              pow(x, p + 1) * pow(y, q + 1) * imgIn[x + (w * y)];
        }
      }
    }
  }
}

void updateTPSVariables(size_t w, size_t h, float *sigma, float *affineParam,
                        float *localCoeff, float *ctrlP) {
  if (sigma != NULL) delete[] sigma;
  if (affineParam != NULL) delete[] affineParam;
  if (localCoeff != NULL) delete[] localCoeff;
  if (ctrlP != NULL) delete[] ctrlP;

  sigma = new float[2];
  affineParam = new float[6];  // affine parameter(a_ij) should be given
  localCoeff =
      new float[2 * w * h];      // the local coefficient(w_ki) should be given
  ctrlP = new float[2 * w * h];  // the control points(c_k) should be given

  // TODO: update the values
}

void tps(float *imgIn, size_t w, size_t h, float *sigma, float *affineParam,
         float *localCoeff, float *ctrlP, float *mmt, int mmtDegree) {
  for (int i = 0; i < 2; i++) {
    sigma[i] = 0;
    for (size_t y = 0; y < h; y++) {
      for (size_t x = 0; x < w; x++) {
        size_t pVector = (i == 0) ? x : y;

        float radialApproximation = radialApprox(w, h, sigma, localCoeff, ctrlP,
                                                 pVector, mmt, mmtDegree, i);
        /**   (a_i1 *x_1)  + (a_i2 *x_2) + a_i3
         *  = (scale*x_1)  + (sheer*x_2) + translation
         *  = (        rotation        ) + translation
         */
        sigma[i] = (affineParam[i * 3] * x) + (affineParam[(i * 3) + 1] * y) +
                   affineParam[(i * 3) + 2] + radialApproximation;
      }
    }
  }
}

float radialApprox(size_t w, size_t h, float *sigma, float *localCoeff,
                   float *ctrlP, size_t pVector, float *mmt, int mmtDegree,
                   int dimIndex) {
  float euclidianDist = 0;
  size_t index;

  size_t dimSize = mmtDegree * mmtDegree;
  sigma[dimIndex] = 0;
  for (size_t i = 0; i < dimSize; i++) {
    index = i + dimSize * dimIndex;
    euclidianDist = pow((ctrlP[index] - pVector), 2) *
                    log(pow((ctrlP[index] - pVector), 2));
    sigma[dimIndex] += localCoeff[index] * euclidianDist;
  }

  return sigma[dimIndex];
}

int pointInPolygon(int nVert, float *vertX, float *vertY, float testX,
                   float testY) {
  /** how we can use???????????????????????????????????????????????*/
  int i, j, c = 0;
  for (i = 0, j = nVert - 1; i < nVert; j = i++) {
    if (((vertY[i] > testY) != (vertY[j] > testY)) &&
        (testX <
         (vertX[j] - vertX[i]) * (testY - vertY[i]) / (vertY[j] - vertY[i]) +
             vertX[i]))
      c = !c;
  }
  return c;
}
