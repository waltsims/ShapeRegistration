#include "hip/hip_runtime.h"
/**
 *  \brief     Shape Registration code
 *  \details   Functions for Nonlinear Shape Registration without
 * Correspondences
 *  \author    Gerasimos Chourdakis
 *  \author    Sungjae Jung
 *  \author    Walter Simson
 *  \version   1.0cse
 *  \date      Mar. 2016
 *  \pre       to be added
 *  \bug       to be added
 *  \warning   to be added
 *  \copyright to be added
 */

#include "helper.h"
#include "shapeRegistrationGPU.h"
#include <stdio.h>

__global__ void setPixelCoordsKernel(PixelCoords *d_pCoords, int d_w, int d_h) {
  int index;

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;

  if (x < d_w && y < d_h) {
    index = x + y * d_w;
    d_pCoords[index].x = (float)y;
    d_pCoords[index].y = (float)x;
  }
}

void setPixelCoordsGPU(PixelCoords *h_pCoords, int h_w, int h_h) {
  dim3 block = dim3(128, 1, 1);
  dim3 grid =
      dim3((h_w + block.x - 1) / block.x, (h_h + block.y - 1) / block.y, 1);

  PixelCoords *d_pCoords;
  hipMalloc(&d_pCoords, h_w * h_h * sizeof(PixelCoords));
  CUDA_CHECK;

  hipMemcpy(d_pCoords, h_pCoords, h_w * h_h * sizeof(PixelCoords),
             hipMemcpyHostToDevice);
  CUDA_CHECK;

  setPixelCoordsKernel << <grid, block>>> (d_pCoords, h_w, h_h);

  hipMemcpy(h_pCoords, d_pCoords, h_w * h_h * sizeof(PixelCoords),
             hipMemcpyDeviceToHost);
  CUDA_CHECK;

  hipFree(d_pCoords);
  CUDA_CHECK;
}

__global__ void imageMomentKernel(float *d_imgIn, PixelCoords *d_pImg, int d_w,
                                  int d_h, float *d_mmt, int d_mmtDegree) {
  int index;

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;

  if (x < d_w && y < d_h) {
    index = x + y * d_w;
    // Compute all the combinations of the (p+q)-order image moments
    // Keep in mind that p,q go from 0 to mmtDegree-1.

    for (int q = 0; q < d_mmtDegree; q++) {
      for (int p = 0; p < d_mmtDegree; p++) {
        int mmtIndex = p + q * d_mmtDegree;

        // Compute the image moments taking the contributions from all the
        // pixels

        d_mmt[mmtIndex * (d_w * d_h) + index] = pow(d_pImg[index].x, p + 1) *
                                                pow(d_pImg[index].y, q + 1) *
                                                d_imgIn[index];
      }
    }
  }
}

void imageMomentGPU(float *h_imgIn, PixelCoords *h_pImg, int h_w, int h_h,
                    float *h_mmt, int h_mmtDegree) {
  dim3 block = dim3(128, 1, 1);
  dim3 grid =
      dim3((h_w + block.x - 1) / block.x, (h_h + block.y - 1) / block.y, 1);

  PixelCoords *d_pImg;
  hipMalloc(&d_pImg, h_w * h_h * sizeof(PixelCoords));
  CUDA_CHECK;

  float *d_imgIn;
  hipMalloc(&d_imgIn, h_w * h_h * sizeof(float));
  CUDA_CHECK;

  float *d_mmt;
  hipMalloc(&d_mmt, h_mmtDegree * h_mmtDegree * h_w * h_h * sizeof(float));
  CUDA_CHECK;

  hipMemcpy(d_imgIn, h_imgIn, h_w * h_h * sizeof(float),
             hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMemcpy(d_pImg, h_pImg, h_w * h_h * sizeof(PixelCoords),
             hipMemcpyHostToDevice);
  CUDA_CHECK;

  imageMomentKernel << <grid, block>>>
      (d_imgIn, d_pImg, h_w, h_h, d_mmt, h_mmtDegree);

  hipMemcpy(h_mmt, d_mmt,
             h_mmtDegree * h_mmtDegree * h_w * h_h * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_imgIn);
  CUDA_CHECK;
  hipFree(d_mmt);
  CUDA_CHECK;
  hipFree(d_pImg);
  CUDA_CHECK;
}

__global__ void setQuadCoordsKernel(QuadCoords *d_qCoords, int d_w, int d_h) {
  int index;

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;

  if (x < d_w && y < d_h) {
    index = x + y * d_w;
    d_qCoords[index].x[0] = (float)x - 0.5;
    d_qCoords[index].y[0] = (float)y - 0.5;
    d_qCoords[index].x[1] = (float)x + 0.5;
    d_qCoords[index].y[1] = (float)y - 0.5;
    d_qCoords[index].x[2] = (float)x + 0.5;
    d_qCoords[index].y[2] = (float)y + 0.5;
    d_qCoords[index].x[3] = (float)x - 0.5;
    d_qCoords[index].y[3] = (float)y + 0.5;
  }
}

void setQuadCoordsGPU(QuadCoords *h_qCoords, int h_w, int h_h) {
  dim3 block = dim3(128, 1, 1);
  dim3 grid =
      dim3((h_w + block.x - 1) / block.x, (h_h + block.y - 1) / block.y, 1);

  QuadCoords *d_qCoords;
  hipMalloc(&d_qCoords, h_w * h_h * sizeof(QuadCoords));
  CUDA_CHECK;

  hipMemcpy(d_qCoords, h_qCoords, h_w * h_h * sizeof(QuadCoords),
             hipMemcpyHostToDevice);
  CUDA_CHECK;

  setQuadCoordsKernel << <grid, block>>> (d_qCoords, h_w, h_h);

  hipMemcpy(h_qCoords, d_qCoords, h_w * h_h * sizeof(QuadCoords),
             hipMemcpyDeviceToHost);

  hipFree(d_qCoords);
  CUDA_CHECK;
}

void cutMarginsGPU(float *imgIn, int w, int h, float *&resizedImg,
                   int &resizedW, int &resizedH, Margins &margins) {
  /** Initialize the the margin positions */
  margins.top = -1;
  margins.bottom = -1;
  margins.left = -1;
  margins.right = -1;

  /** Top: row (y) of the first foreground pixel from top. */
  for (int y = 0; y < h; y++) {
    for (int x = 0; x < w; x++) {
      if (imgIn[x + (w * y)] == FOREGROUND) {
        margins.top = y;
        break;
      }
    }
    if (margins.top != -1) {
      break;
    }
  }

  /** Bottom: row (y) of the last foreground pixel from top. */
  for (int y = h - 1; y >= 0; y--) {
    for (int x = 0; x < w; x++) {
      if (imgIn[x + (w * y)] == FOREGROUND) {
        margins.bottom = y;
        break;
      }
    }
    if (margins.bottom != -1) {
      break;
    }
  }

  /** Left: column (x) of the first foreground pixel from left. */
  for (int x = 0; x < w; x++) {
    for (int y = 0; y < h; y++) {
      if (imgIn[x + (w * y)] == FOREGROUND) {
        margins.left = x;
        break;
      }
    }
    if (margins.left != -1) {
      break;
    }
  }

  /** Right: column (x) of the last foreground pixel from left. */
  for (int x = w - 1; x >= 0; x--) {
    for (int y = 0; y < h; y++) {
      if (imgIn[x + (w * y)] == FOREGROUND) {
        margins.right = x;
        break;
      }
    }
    if (margins.right != -1) {
      break;
    }
  }

  /** Height and width of the cropped image */
  resizedH = margins.bottom - margins.top + 1;
  resizedW = margins.right - margins.left + 1;

  /** Allocate the cropped image array */
  resizedImg = new float[resizedW * resizedH];

  /** Assign the respective full image pixels to the cropped image pixels */
  for (int y = 0; y < resizedH; y++) {
    for (int x = 0; x < resizedW; x++) {
      resizedImg[x + resizedW * y] =
          imgIn[(x + margins.left) + (w * (y + margins.top))];
    }
  }
}

__global__ void addMarginsKernel(float *d_resizedImg, int d_resizedW,
                                 int d_resizedH, float *d_imgOut, int d_w,
                                 int d_h, Margins &d_margins) {
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  if (x < d_w && y < d_h) {
    int index = x + y * d_w;
    int res_index = (x - d_margins.left) + (y - d_margins.top) * d_resizedW;

    if (x >= d_margins.left && x <= d_margins.right && y >= d_margins.top &&
        y <= d_margins.bottom) {
      d_imgOut[index] = d_resizedImg[res_index];
    } else {
      d_imgOut[index] = BACKGROUND;
    }
  }
}

void addMarginsGPU(float *resizedImg, int resizedW, int resizedH, float *imgOut,
                   int w, int h, Margins &margins) {
  for (int y = 0; y < h; y++) {
    for (int x = 0; x < w; x++) {
      int index = x + y * w;
      int res_index = (x - margins.left) + (y - margins.top) * resizedW;
      if (x >= margins.left && x <= margins.right && y >= margins.top &&
          y <= margins.bottom) {
        imgOut[index] = resizedImg[res_index];
      } else {
        imgOut[index] = BACKGROUND;
      }
    }
  }
}

__global__ void centerOfMassKernel(float *d_imgIn, int d_w, int d_h,
                                   float *d_xCentCoord, float *d_yCentCoord,
                                   float *numberOfForegroundPixel) {
  int index;

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int x_t = threadIdx.x;
  int y_d = blockIdx.x;

  extern __shared__ float s_data[];

  if (x_t < d_w && y_d < d_h) {
    s_data[x_t] = 0.0;
    s_data[x_t + d_w] = 0.0;
    s_data[x_t + d_w + d_w] = 0.0;
  }

  __syncthreads();

  if (x_t < d_w && y_d < d_h) {
    index = x_t + d_w * y_d;
    if (d_imgIn[index] == FOREGROUND) {
      atomicAdd(&s_data[x_t], float(x_t));
      atomicAdd(&s_data[x_t + d_w], float(y_d));
      atomicAdd(&s_data[x_t + d_w + d_w], 1.0f);
    }
  }

  __syncthreads();
  if (x_t == 0 && y_d == 1) {
    printf("check3\n");
  }
  if (x_t < d_w && y_d < d_h) {
    for (int offset = d_w / 2; offset > 0; offset /= 2) {
      if (x_t < offset) {
        s_data[x_t] += s_data[x_t + offset];
        s_data[x_t + d_w] += s_data[x_t + d_w + offset];
        s_data[x_t + d_w + d_w] += s_data[x_t + d_w + d_w + offset];
      }
      __syncthreads();
    }
  }

  __syncthreads();
  if (x_t == 0 && y_d == 1) {
    printf("check4\n");
  }

  if (x_t == 0 && y_d == 1) {
    printf("check5\n");
  }

  if (x_t == 0 && y_d < d_h) {
    d_xCentCoord[y_d] = s_data[0];
    d_yCentCoord[y_d] = s_data[d_w];
    numberOfForegroundPixel[y_d] = s_data[d_w + d_w];
  }
  if (x_t == 0 && y_d == 1) {
    printf("check6\n");
  }
}

void centerOfMassGPU(float *h_imgIn, int h_w, int h_h, float &h_xCentCoord,
                     float &h_yCentCoord) {
  const int sizeOfImg = h_w * h_h;
  const int nThreads = h_w;
  const int sizeOfReduction = (sizeOfImg / nThreads + sizeOfImg % nThreads);

  printf("h_w, h_h : (%d, %d)\n", h_w, h_h);

  dim3 block = dim3(nThreads, 1, 1);
  dim3 grid = dim3(h_w, 1, 1);

  float *numOfForegroundPixelOut = new float[h_w * sizeof(float)];
  float *xCentCoordsOut = new float[h_w * sizeof(float)];
  float *yCentCoordsOut = new float[h_w * sizeof(float)];

  float *d_imgIn;
  float *d_xCentCoord;
  float *d_yCentCoord;
  float *d_numOfForegroundPixelOut;

  hipMalloc(&d_imgIn, h_w * h_h * sizeof(float));
  CUDA_CHECK;
  hipMalloc(&d_xCentCoord, h_w * sizeof(float));
  CUDA_CHECK;
  hipMalloc(&d_yCentCoord, h_h * sizeof(float));
  CUDA_CHECK;
  hipMalloc(&d_numOfForegroundPixelOut, h_h * sizeof(float));
  CUDA_CHECK;

  hipMemcpy(d_imgIn, h_imgIn, h_w * h_h * sizeof(float),
             hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMemset(d_xCentCoord, 0, h_w * sizeof(float));
  CUDA_CHECK;
  hipMemset(d_yCentCoord, 0, h_h * sizeof(float));
  CUDA_CHECK;
  hipMemset(d_numOfForegroundPixelOut, 0, h_h * sizeof(float));
  CUDA_CHECK;

  centerOfMassKernel << <grid, block, 3 * nThreads * sizeof(float)>>>
      (d_imgIn, h_w, h_h, d_xCentCoord, d_yCentCoord,
       d_numOfForegroundPixelOut);
  CUDA_CHECK;

  hipMemcpy(xCentCoordsOut, d_xCentCoord, h_w * sizeof(float),
             hipMemcpyDeviceToHost);
  CUDA_CHECK;
  hipMemcpy(yCentCoordsOut, d_yCentCoord, h_w * sizeof(float),
             hipMemcpyDeviceToHost);
  CUDA_CHECK;
  hipMemcpy(numOfForegroundPixelOut, d_numOfForegroundPixelOut,
             h_w * sizeof(float), hipMemcpyDeviceToHost);
  CUDA_CHECK;

  h_xCentCoord = 0;
  h_yCentCoord = 0;
  for (int i = 0; i < h_w; i++) {
    h_xCentCoord += xCentCoordsOut[i];
    h_yCentCoord += yCentCoordsOut[i];
    if (i != 0) numOfForegroundPixelOut[0] += numOfForegroundPixelOut[i];
  }

  h_xCentCoord /= numOfForegroundPixelOut[0];
  h_yCentCoord /= numOfForegroundPixelOut[0];

  hipFree(d_imgIn);
  CUDA_CHECK;
  hipFree(d_xCentCoord);
  CUDA_CHECK;
  hipFree(d_yCentCoord);
  CUDA_CHECK;
}

__global__ void pCoordsNormalizationKernel(int d_w, int d_h,
                                           PixelCoords *d_pCoords,
                                           float d_xCentCoord,
                                           float d_yCentCoord) {
  int index;

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;

  float normXFactor = 0.5 / max(d_xCentCoord, d_w - d_xCentCoord);
  float normYFactor = 0.5 / max(d_yCentCoord, d_h - d_yCentCoord);

  if (x < d_w && y < d_h) {
    index = x + y * d_w;

    d_pCoords[index].x = (d_pCoords[index].x - d_xCentCoord) * normXFactor;
    d_pCoords[index].y = (d_pCoords[index].y - d_yCentCoord) * normYFactor;
  }
}

void pCoordsNormalizationGPU(int h_w, int h_h, PixelCoords *h_pCoords,
                             float h_xCentCoord, float h_yCentCoord) {
  dim3 block = dim3(128, 1, 1);
  dim3 grid =
      dim3((h_w + block.x - 1) / block.x, (h_h + block.y - 1) / block.y, 1);

  PixelCoords *d_pCoords;
  hipMalloc(&d_pCoords, h_w * h_h * sizeof(PixelCoords));
  CUDA_CHECK;

  hipMemcpy(d_pCoords, h_pCoords, h_w * h_h * sizeof(PixelCoords),
             hipMemcpyHostToDevice);
  CUDA_CHECK;

  pCoordsNormalizationKernel << <grid, block>>>
      (h_w, h_h, d_pCoords, h_xCentCoord, h_yCentCoord);

  hipMemcpy(h_pCoords, d_pCoords, h_w * h_h * sizeof(PixelCoords),
             hipMemcpyDeviceToHost);
  CUDA_CHECK;

  hipFree(d_pCoords);
  CUDA_CHECK;
}

__global__ void qCoordsNormalizationKernel(int d_w, int d_h,
                                           QuadCoords *d_qCoords,
                                           float d_xCentCoord,
                                           float d_yCentCoord) {
  int index;

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;

  float normXFactor = 0.5 / max(d_xCentCoord, d_w - d_xCentCoord);
  float normYFactor = 0.5 / max(d_yCentCoord, d_h - d_yCentCoord);

  if (x < d_w && y < d_h) {
    index = x + y * d_w;
    d_qCoords[index].x[0] =
        (d_qCoords[index].x[0] - d_xCentCoord) * normXFactor;
    d_qCoords[index].y[0] =
        (d_qCoords[index].y[0] - d_yCentCoord) * normYFactor;
    d_qCoords[index].x[1] =
        (d_qCoords[index].x[1] - d_xCentCoord) * normXFactor;
    d_qCoords[index].y[1] =
        (d_qCoords[index].y[1] - d_yCentCoord) * normYFactor;
    d_qCoords[index].x[2] =
        (d_qCoords[index].x[2] - d_xCentCoord) * normXFactor;
    d_qCoords[index].y[2] =
        (d_qCoords[index].y[2] - d_yCentCoord) * normYFactor;
    d_qCoords[index].x[3] =
        (d_qCoords[index].x[3] - d_xCentCoord) * normXFactor;
    d_qCoords[index].y[3] =
        (d_qCoords[index].y[3] - d_yCentCoord) * normYFactor;
  }
}

void qCoordsNormalizationGPU(int h_w, int h_h, QuadCoords *h_qCoords,
                             float h_xCentCoord, float h_yCentCoord) {
  dim3 block = dim3(128, 1, 1);
  dim3 grid =
      dim3((h_w + block.x - 1) / block.x, (h_h + block.y - 1) / block.y, 1);

  QuadCoords *d_qCoords;
  hipMalloc(&d_qCoords, h_w * h_h * sizeof(QuadCoords));
  CUDA_CHECK;

  hipMemcpy(d_qCoords, h_qCoords, h_w * h_h * sizeof(QuadCoords),
             hipMemcpyHostToDevice);
  CUDA_CHECK;

  qCoordsNormalizationKernel << <grid, block>>>
      (h_w, h_h, d_qCoords, h_xCentCoord, h_yCentCoord);

  hipMemcpy(h_qCoords, d_qCoords, h_w * h_h * sizeof(QuadCoords),
             hipMemcpyDeviceToHost);
  CUDA_CHECK;

  hipFree(d_qCoords);
  CUDA_CHECK;
}

__device__ float radialApproxKernel(float d_x, float d_y, float d_cx,
                                    float d_cy) {
  float d_r2 = (d_cx - d_x) * (d_cx - d_x) + (d_cy - d_y) * (d_cy - d_y);

  return d_r2 < 0.0000000001 ? 0 : d_r2 * log(d_r2);
}

__global__ void pTPSGPUKernel(int d_w, int d_h, PixelCoords *d_pCoords,
                              TPSParams d_tpsParams, int d_cDim) {
  int index;

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;

  index = x + d_w * y;
  int dimSize = d_cDim * d_cDim;
  float Q;
  float freeDeformation[2] = {0, 0};

  if (x < d_w && y < d_h) {
    Q = 0;
    freeDeformation[0] = 0;
    freeDeformation[1] = 0;
    // for all c_m support coordinates
    for (int k = 0; k < dimSize; k++) {
      // calculate radial approximation

      Q = radialApproxKernel(d_pCoords[index].x, d_pCoords[index].y,
                             d_tpsParams.ctrlP[k],
                             d_tpsParams.ctrlP[k + dimSize]);

      // multiply with weights
      for (int i = 0; i < 2; i++) {
        freeDeformation[i] += d_tpsParams.localCoeff[k + (i * dimSize)] * Q;
      }
    }

    // note:: change
    float tempQCoordsX = d_pCoords[index].x;
    float tempQCoordsY = d_pCoords[index].y;

    d_pCoords[index].x = (d_tpsParams.affineParam[0] * tempQCoordsX) +
                         (d_tpsParams.affineParam[1] * tempQCoordsY) +
                         d_tpsParams.affineParam[2] + freeDeformation[0];

    d_pCoords[index].y = (d_tpsParams.affineParam[3] * tempQCoordsX) +
                         (d_tpsParams.affineParam[4] * tempQCoordsY) +
                         d_tpsParams.affineParam[5] + freeDeformation[1];
  }
}

void pTPSGPU(int h_w, int h_h, PixelCoords *h_pCoords, TPSParams &h_tpsParams,
             int h_cDim) {
  dim3 block = dim3(128, 1, 1);
  dim3 grid =
      dim3((h_w + block.x - 1) / block.x, (h_h + block.y - 1) / block.y, 1);

  PixelCoords *d_pCoords;
  hipMalloc(&d_pCoords, h_h * h_w * sizeof(PixelCoords));
  CUDA_CHECK;

  hipMemcpy(d_pCoords, h_pCoords, h_h * h_w * sizeof(PixelCoords),
             hipMemcpyHostToDevice);
  CUDA_CHECK;

  pTPSGPUKernel << <grid, block>>> (h_w, h_h, d_pCoords, h_tpsParams, h_cDim);
  CUDA_CHECK;

  hipMemcpy(h_pCoords, d_pCoords, h_h * h_w * sizeof(PixelCoords),
             hipMemcpyDeviceToHost);
  CUDA_CHECK;

  hipFree(d_pCoords);
  CUDA_CHECK;
}

__global__ void qTPSKernel(int d_w, int d_h, QuadCoords *d_qCoords,
                           TPSParams d_tpsParams, int d_cDim) {
  int index;

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;

  index = x + d_w * y;
  int dimSize = d_cDim * d_cDim;
  float Q;
  float freeDeformation[2] = {0, 0};

  if (x < d_w && y < d_h) {
    for (int qIndex = 0; qIndex < 4; qIndex++) {
      Q = 0;
      freeDeformation[0] = 0;
      freeDeformation[1] = 0;
      // for all c_m support coordinates
      for (int k = 0; k < dimSize; k++) {
        // calculate radial approximation

        Q = radialApproxKernel(d_qCoords[index].x[qIndex],
                               d_qCoords[index].y[qIndex], d_tpsParams.ctrlP[k],
                               d_tpsParams.ctrlP[k + dimSize]);

        // multiply with weights
        for (int i = 0; i < 2; i++) {
          freeDeformation[i] += d_tpsParams.localCoeff[k + (i * dimSize)] * Q;
        }
      }

      // note:: change
      float tempQCoordsX = d_qCoords[index].x[qIndex];
      float tempQCoordsY = d_qCoords[index].y[qIndex];

      d_qCoords[index].x[qIndex] = (d_tpsParams.affineParam[0] * tempQCoordsX) +
                                   (d_tpsParams.affineParam[1] * tempQCoordsY) +
                                   d_tpsParams.affineParam[2] +
                                   freeDeformation[0];

      d_qCoords[index].y[qIndex] = (d_tpsParams.affineParam[3] * tempQCoordsX) +
                                   (d_tpsParams.affineParam[4] * tempQCoordsY) +
                                   d_tpsParams.affineParam[5] +
                                   freeDeformation[1];
    }
  }
}

void qTPSGPU(int h_w, int h_h, QuadCoords *h_qCoords, TPSParams &h_tpsParams,
             int h_cDim) {
  dim3 block = dim3(128, 1, 1);
  dim3 grid =
      dim3((h_w + block.x - 1) / block.x, (h_h + block.y - 1) / block.y, 1);

  QuadCoords *d_qCoords;
  hipMalloc(&d_qCoords, h_h * h_w * sizeof(QuadCoords));
  CUDA_CHECK;

  hipMemcpy(d_qCoords, h_qCoords, h_h * h_w * sizeof(QuadCoords),
             hipMemcpyHostToDevice);
  CUDA_CHECK;

  qTPSKernel <<< grid, block >>> (h_w, h_h, d_qCoords, h_tpsParams, h_cDim);
  CUDA_CHECK;

  hipMemcpy(h_qCoords, d_qCoords, h_h * h_w * sizeof(QuadCoords),
             hipMemcpyDeviceToHost);
  CUDA_CHECK;

  hipFree(d_qCoords);
  CUDA_CHECK;
}

__global__ void jacobianTransGPUKernel(int d_w, int d_h, float *d_jacobi,
                                        TPSParams &d_tpsParams, int d_c_dim) {

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    int indexP, indexJ;
    int K = d_c_dim * d_c_dim;
    float squareOfNorm;
    float precomp;
    float x_j;

    if (x < d_w && y < d_h) {

      indexP = x + d_w * y;

      for (int i = 0; i < 2; i++) {
        for (int j = 0; j < 2; j++) {
          indexJ = 4 * indexP + i + 2 * j;
          d_jacobi[indexJ] = d_tpsParams.affineParam[i + 2 * j];
        }
      }

      __syncthreads();

      for (int k = 0; k < K; k++) {
        squareOfNorm =
            (d_tpsParams.ctrlP[k] - x) * (d_tpsParams.ctrlP[k] - x) +
            (d_tpsParams.ctrlP[k + K] - y) * (d_tpsParams.ctrlP[k + K] - y);

        if (squareOfNorm > 0.000001) {
          precomp = 2 * (1 + log(squareOfNorm));
        } else {
          precomp = 2;
        }

        for (int i = 0; i < 2; i++) {
          for (int j = 0; j < 2; j++) {
            indexJ = 4 * indexP + i + 2 * j;
            x_j = (j == 0 ? x : y);
            d_jacobi[indexJ] -= precomp * d_tpsParams.localCoeff[k + i * K] *
                              (d_tpsParams.ctrlP[k + j * K] - x_j);
          }
        }
      }
    }
}

void jacobianTransGPU(int h_w, int h_h, float *h_jacobi, TPSParams h_tpsParams,
                      int h_c_dim) {
  dim3 block = dim3(128, 1, 1);
  dim3 grid =
      dim3((h_w + block.x - 1) / block.x, (h_h + block.y - 1) / block.y, 1);

  float *d_jacobi;
  hipMalloc(&d_jacobi, h_w * h_h * 4 * sizeof(float));
  CUDA_CHECK;
  hipMemset(d_jacobi, 0, h_w * h_h * 4 * sizeof(float));
  CUDA_CHECK;

  jacobianTransGPUKernel <<<grid, block>>>
      (h_w, h_h, d_jacobi, h_tpsParams, h_c_dim);
  CUDA_CHECK;

  hipMemcpy(h_jacobi, d_jacobi, h_h * h_w * 4 * sizeof(float),
             hipMemcpyDeviceToHost);
  CUDA_CHECK;

  hipFree(d_jacobi);
  CUDA_CHECK;
}

/*  int index;
  int p_index;

  for (int j = 0; j < t_h; j++) {
    for (int i = 0; i < t_w; i++) {
      index = i + t_w * j;
      if (imgIn[index] == FOREGROUND) {
        float xpolygon[4] = {qCoords[index].x[0], qCoords[index].x[1],
                             qCoords[index].x[2], qCoords[index].x[3]};
        float ypolygon[4] = {qCoords[index].y[0], qCoords[index].y[1],
                             qCoords[index].y[2], qCoords[index].y[3]};



        int xLeftOffset = int(floor(min(qCoords[index].x[0],
  qCoords[index].x[3])));
        int xRightOffset = int(ceil(max(qCoords[index].x[1],
  qCoords[index].x[2])));
        int yTopOffset = int(floor(min(qCoords[index].y[0],
  qCoords[index].y[1])));
        int yBottomOffset = int(ceil(max(qCoords[index].y[2],
  qCoords[index].y[3])));


        printf("------------i, j = %d, %d-------------\n", i, j);
        printf("left, right = %d, %d||", xLeftOffset, xRightOffset);
        printf("top, bottom = %d, %d\n", yTopOffset, yBottomOffset);
        // TODO create local index to search for neignboring points
        // withing bounding box of polygon
        for (int y = yTopOffset; y < yBottomOffset; y++) {
          for (int x = xLeftOffset; x < xRightOffset; x++) {
            p_index = x + o_w * y;

            if (pointInPolygonGPU(4, xpolygon, ypolygon, pCoords[p_index].x,
                               pCoords[p_index].y))
              imgOut[p_index] = FOREGROUND;
          }
        }
      }
    }
  }*/

__device__ bool pointInPolygonKernel(int nVert, float *vertX, float *vertY,
                                     float testX, float testY) {
  int i, j;
  bool c = false;

  for (i = 0, j = nVert - 1; i < nVert; j = i++) {
    if (((vertY[i] > testY) != (vertY[j] > testY)) &&
        (testX <
         (vertX[j] - vertX[i]) * (testY - vertY[i]) / (vertY[j] - vertY[i]) +
             vertX[i]))
      c = !c;
  }

  return c;
}

__global__ void transferKernel(float *d_imgIn, PixelCoords *d_pCoords,
                               QuadCoords *d_qCoords, int d_t_w, int d_t_h,
                               int d_o_w, int d_o_h, float *d_imgOut) {
  int index;

  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;

  int p_index;

  if (i < d_t_w && j < d_t_h) {
    index = i + d_t_w * j;
    if (d_imgIn[index] == FOREGROUND) {
      float xpolygon[4] = {d_qCoords[index].x[0], d_qCoords[index].x[1],
                           d_qCoords[index].x[2], d_qCoords[index].x[3]};
      float ypolygon[4] = {d_qCoords[index].y[0], d_qCoords[index].y[1],
                           d_qCoords[index].y[2], d_qCoords[index].y[3]};

      for (int x = 0; x < d_o_w; x++) {
        for (int y = 0; y < d_o_h; y++) {
          p_index = x + d_o_w * y;

          if (pointInPolygonKernel(4, xpolygon, ypolygon, d_pCoords[p_index].x,
                                   d_pCoords[p_index].y))
            d_imgOut[p_index] = FOREGROUND;
        }
      }
    }
  }
}

void transferGPU(float *h_imgIn, PixelCoords *h_pCoords, QuadCoords *h_qCoords,
                 int h_t_w, int h_t_h, int h_o_w, int h_o_h, float *h_imgOut) {
  dim3 block = dim3(128, 1, 1);
  dim3 grid =
      dim3((h_t_w + block.x - 1) / block.x, (h_t_h + block.y - 1) / block.y, 1);

  float *d_imgIn;
  PixelCoords *d_pCoords;
  QuadCoords *d_qCoords;
  float *d_imgOut;

  hipMalloc(&d_imgIn, h_t_w * h_t_h * sizeof(float));
  CUDA_CHECK;
  hipMalloc(&d_pCoords, h_o_w * h_o_h * sizeof(PixelCoords));
  CUDA_CHECK;
  hipMalloc(&d_qCoords, h_t_w * h_t_h * sizeof(QuadCoords));
  CUDA_CHECK;
  hipMalloc(&d_imgOut, h_o_w * h_o_h * sizeof(float));
  CUDA_CHECK;

  hipMemcpy(d_imgIn, h_imgIn, h_t_w * h_t_h * sizeof(float),
             hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMemcpy(d_pCoords, h_pCoords, h_o_w * h_o_h * sizeof(PixelCoords),
             hipMemcpyHostToDevice);
  CUDA_CHECK;
  hipMemcpy(d_qCoords, h_qCoords, h_t_w * h_t_h * sizeof(QuadCoords),
             hipMemcpyHostToDevice);
  CUDA_CHECK;

  hipMemset(d_imgOut, 0, h_o_w * h_o_h * sizeof(float));
  CUDA_CHECK;

  transferKernel << <grid, block>>>
      (d_imgIn, d_pCoords, d_qCoords, h_t_w, h_t_h, h_o_w, h_o_h, d_imgOut);

  /*  hipMemcpy(h_imgIn, d_imgIn, h_t_w * h_t_h * sizeof(float),
               hipMemcpyDeviceToHost);
    CUDA_CHECK;
    hipMemcpy(h_pCoords, d_pCoords, h_o_w * h_o_h * sizeof(PixelCoords),
               hipMemcpyDeviceToHost);
    CUDA_CHECK;
    hipMemcpy(h_qCoords, d_qCoords, h_t_w * h_t_h * sizeof(QuadCoords),
               hipMemcpyDeviceToHost);
    CUDA_CHECK;*/
  hipMemcpy(h_imgOut, d_imgOut, h_o_w * h_o_h * sizeof(float),
             hipMemcpyDeviceToHost);
  CUDA_CHECK;

  hipFree(d_imgIn);
  CUDA_CHECK;
  hipFree(d_pCoords);
  CUDA_CHECK;
  hipFree(d_qCoords);
  CUDA_CHECK;
  hipFree(d_imgOut);
  CUDA_CHECK;
}
/*
__global__ void objectiveFunctionGPU(
    float *d_observationImg, float *d_templateImg, float *d_jacobi, int d_ow,
    int d_oh, double *d_normalisation, TPSParams d_tpsParams,
    QuadCoords *d_qTemplate, PixelCoords *d_pTemplate,
    PixelCoords *d_pObservation, int d_tw, int rt_h, float *d_residual) {
  residual[index] =
      (sumObsMoment[index] - sumTempMoment[index]) / normalisation[index];
}

void objectiveFunctionGPU(float *observationImg, float *templateImg,
                          float *jacobi, int ro_w, int ro_h,
                          double *normalisation, TPSParams &tpsParams,
                          QuadCoords *qTemplate, PixelCoords *pTemplate,
                          PixelCoords *pObservation, int rt_w, int rt_h,
                          float *residual) {
  int momentDeg = 9;

  float *observationMoment = new float[momentDeg * momentDeg * ro_w * ro_h];
  float *templateMoment = new float[momentDeg * momentDeg * rt_w * rt_h];

  float sumTempMoment[momentDeg * momentDeg];
  float sumObsMoment[momentDeg * momentDeg];
  // init moment array
  for (int init = 0; init < momentDeg * momentDeg; init++) {
    sumObsMoment[init] = (float)0;
    sumTempMoment[init] = (float)0;
  }

  qTPSKernel << <grid, block>>> (d_tw, d_th, d_qTemplate, d_tpsParams, d_cDim);

  transferKernel << <grid, block>>> (d_templateImg, d_pObservation, d_qTemplate,
                                     d_ow, d_th, d_ow, d_oh, d_imgOut);

  // TODO how and when to allocate memory for observation moments and moment
  // deg.
  imageMomentKernel << <grid, block>>> (d_observationImg, d_pObservation, d_ow,
                                        d_oh, d_observationMoment, d_mmtDegree);
  imageMomentKernel << <grid, block>>>
      (d_templateImg, d_pTemplate, d_tw, d_th, d_templateMoment, d_mmtDegree);

  // TODO call jacobian kernel here

  // get jacobian of current tps params
  jacobianTrans(rt_w, rt_h, jacobi, pTemplate, tpsParams, DIM_C_REF);
  // get determinant of Jacobian

  // TODO two reduces needed here
  for (int index = 0; index < momentDeg * momentDeg; index++) {
    for (int y = 0; y < rt_h; y++) {
      for (int x = 0; x < rt_w; x++) {
        sumTempMoment[index] +=
            templateMoment[index * (rt_h * rt_w) + (x + rt_w * y)] *
            jacobi[x + rt_w * y];
      }
    }

    for (int y = 0; y < ro_h; y++) {
      for (int x = 0; x < ro_w; x++) {
        sumObsMoment[index] +=
            observationMoment[index * (ro_h * ro_w) + (x + ro_w * y)];
      }
    }

    residual[index] =
        (sumObsMoment[index] - sumTempMoment[index]) / normalisation[index];
  }
  delete[] observationMoment;
  delete[] templateMoment;
};*/